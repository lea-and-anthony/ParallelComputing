#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdint.h>
#include <iostream>
#include "kernel.h"

void cudaDisplayError(char *functionName, hipError_t cudaStatus)
{
	std::cerr << std::endl << functionName << " failed!" << std::endl << hipGetErrorString(cudaStatus) << std::endl;
}

bool transferMemory(void** dest, void* src, size_t size)
{
	hipError_t cudaStatus;
	cudaStatus = hipMalloc(dest, size);
	if (cudaStatus != hipSuccess)
	{
		cudaDisplayError("hipMalloc", cudaStatus);
		return false;
	}
	cudaStatus = hipMemcpy(*dest, src, size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		cudaDisplayError("hipMemcpy", cudaStatus);
		hipFree(dest);
		return false;
	}
	return true;
}

void startKernel(Sample<FeatureType> &sample, NodeGPU *tree, uint32_t treeSize, uint32_t *histograms, uint32_t histSize, FeatureType *features, uint32_t featuresSize, int16_t height, int16_t width, FeatureType *features_integral, uint32_t featuresIntegralSize, int16_t height_integral, int16_t width_integral, size_t numLabels, int lPXOff, int lPYOff, unsigned int *out_result)
{
	bool success = false;

	// Memory transfer for tree
	NodeGPU *treeGPU = NULL;
	success = transferMemory((void**)&treeGPU, (void*)tree, treeSize*sizeof(NodeGPU));
	if (!success)
	{
		return;
	}

	// Memory transfer for histograms
	uint32_t *histogramsGPU = NULL;
	success = transferMemory((void**)&histogramsGPU, (void*)histograms, histSize*sizeof(uint32_t));
	if (!success)
	{
		hipFree(treeGPU);
		return;
	}

	// Memory transfer for features
	FeatureType *featuresGPU = NULL;
	success = transferMemory((void**)&featuresGPU, (void*)features, featuresSize*sizeof(FeatureType));
	if (!success)
	{
		hipFree(treeGPU);
		hipFree(histogramsGPU);
		return;
	}

	// Memory transfer for features_integral
	FeatureType *features_integralGPU = NULL;
	success = transferMemory((void**)&features_integralGPU, (void*)features_integral, featuresIntegralSize*sizeof(FeatureType));
	if (!success)
	{
		hipFree(treeGPU);
		hipFree(histogramsGPU);
		hipFree(featuresGPU);
		return;
	}
	
	// Memory transfer for out_result
	unsigned int *out_resultGPU = NULL;
	success = transferMemory((void**)&out_resultGPU, (void*)out_result, width*height*numLabels*sizeof(unsigned int));
	if (!success)
	{
		hipFree(treeGPU);
		hipFree(histogramsGPU);
		hipFree(featuresGPU);
		hipFree(features_integralGPU);
		return;
	}

	// Kernel launch
	const int SIZE_BLOCK = 32;
	dim3 dimBlock(SIZE_BLOCK, SIZE_BLOCK);
	dim3 dimGrid((int)ceil(width * 1.0f / SIZE_BLOCK), (int)ceil(height * 1.0f / SIZE_BLOCK));
	kernel << <dimGrid, dimBlock >> > (sample, treeGPU, histogramsGPU, featuresGPU, features_integralGPU, height, width, height_integral, width_integral, numLabels, lPXOff, lPYOff, out_resultGPU);
	hipDeviceSynchronize();

	// Kernel end
	hipError_t cudaStatus = hipGetLastError();
	hipFree(treeGPU);
	hipFree(histogramsGPU);
	hipFree(featuresGPU);
	hipFree(features_integralGPU);

	if (cudaStatus != hipSuccess)
	{
		cudaDisplayError("hipGetLastError", cudaStatus);
		hipFree(out_resultGPU);
		return;
	}

	// Memory transfer
	cudaStatus = hipMemcpy(out_result, out_resultGPU, width*height*numLabels*sizeof(unsigned int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		cudaDisplayError("hipMemcpy", cudaStatus);
		hipFree(out_resultGPU);
		return;
	}
	hipFree(out_resultGPU);

	return;
}

__global__ void kernel(Sample<FeatureType> sample, NodeGPU *tree, uint32_t *histograms, FeatureType *features, FeatureType *features_integral, int16_t height, int16_t width, int16_t height_integral, int16_t width_integral, size_t numLabels, int lPXOff, int lPYOff, unsigned int *out_result)
{
	sample.x = blockIdx.x*blockDim.x + threadIdx.x;
	if (sample.x >= width)
	{
		return;
	}
	sample.y = blockIdx.y*blockDim.y + threadIdx.y;
	if (sample.y >= height)
	{
		return;
	}

	uint32_t histIterator = predictNoPtr(sample, tree, histograms, features, features_integral, height, width, height_integral, width_integral);

	for (int y = (int)sample.y - lPYOff; y <= (int)sample.y + lPYOff; ++y)
	{
		for (int x = (int)sample.x - lPXOff; x <= (int)sample.x + lPXOff; ++x, ++histIterator)
		{
			if (histograms[histIterator] >= numLabels)
			{
				//std::cerr << "Invalid label in prediction: " << histograms[histIterator] << "\n";
				asm("trap;");
			}

			if (x >= 0 && x < width && y >= 0 && y < height)
			{
				atomicAdd(out_result + (histograms[histIterator] * height * width + y * width + x), 1);
				// out_result[histograms[histIterator] * height * width + y * width + x]++;
			}
		}
	}
}
